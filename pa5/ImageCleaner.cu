#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------

#define PI	3.14159256
#define DIRECTION_X 1
#define DIRECTION_Y 1

__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
    // Currently does nothing
}


__global__ void fftKernel(float *real_image, float *imag_image, int direction)
{
    // thread block represents a single x
    // each thread in block is computing a particular y

    int row_idx = threadIdx.x;
    int image_index;
    if (direction == DIRECTION_X) {
        image_index = blockIdx.x*SIZEY + threadIdx.x;
    } else {
        image_index = threadIdx.x*SIZEY + blockIdx.x;
    }
    // allocate row y
    __shared__ float real_row[SIZEY];
    __shared__ float imag_row[SIZEY];

    real_row[row_idx] = real_image[image_index];
    imag_row[row_idx] = real_image[image_index];

    __syncthreads();

    // Compute the value for this index
    float real_value = 0;
    float imag_value = 0;
    for(unsigned int n = 0; n < SIZEY; n++)
    {
        float term = -2 * PI * row_idx * n / SIZEY;
        float real_term = cos(term);
        float imag_term = sin(term);
        real_value += (real_row[n] * real_term) - (imag_row[n] * imag_term);
        imag_value += (imag_row[n] * real_term) + (real_row[n] * imag_term);
    }

    real_image[image_index] = real_value;
    imag_image[image_index] = imag_value;
}

__global__ void ifftKernel(float *real_image, float *imag_image, int direction)
{
    // thread block represents a single x
    // each thread in block is computing a particular y

    int row_idx = threadIdx.x;
    int image_idx;
    if (direction == DIRECTION_X) {
        image_index = blockIdx.x*SIZEY + threadIdx.x;
    } else {
        image_index = threadIdx.x*SIZEY + blockIdx.x;
    }
    // allocate row y
    __shared__ float real_row[SIZEY];
    __shared__ float imag_row[SIZEY];

    real_row[row_idx] = real_image[image_index];
    imag_row[row_idx] = real_image[image_index];

    __syncthreads();

    // Compute the value for this index
    float real_value = 0;
    float imag_value = 0;
    for(unsigned int n = 0; n < SIZEY; n++)
    {
        float term = 2 * PI * row_idx * n / SIZEY;
        float real_term = cos(term);
        float imag_term = sin(term);
        real_value += (real_row[n] * real_term) - (imag_row[n] * imag_term);
        imag_value += (imag_row[n] * real_term) + (real_row[n] * imag_term);
    }

    real_image[image_index] = real_value / SIZEY;
    imag_image[image_index] = imag_value / SIZEY;
}

__global__ void cpu_filter(float *real_image, float *imag_image, int size_x, int size_y)
{
  int eightX = size_x/8;
  int eight7X = size_x - eightX;
  int eightY = size_y/8;
  int eight7Y = size_y - eightY;
  for(unsigned int x = 0; x < size_x; x++)
  {
    for(unsigned int y = 0; y < size_y; y++)
    {
      if(!(x < eightX && y < eightY) &&
	 !(x < eightX && y >= eight7Y) &&
	 !(x >= eight7Y && y < eightY) &&
	 !(x >= eight7Y && y >= eight7Y))
      {
	// Zero out these values
	real_image[y*size_x + x] = 0;
	imag_image[y*size_x + x] = 0;
      }
    }
  }
}



//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
    // check that the sizes match up
    assert(size_x == SIZEX);
    assert(size_y == SIZEY);

    int matSize = size_x * size_y * sizeof(float);

    // These variables are for timing purposes
    float transferDown = 0, transferUp = 0, execution = 0;
    hipEvent_t start,stop;

    CUDA_ERROR_CHECK(hipEventCreate(&start));
    CUDA_ERROR_CHECK(hipEventCreate(&stop));

    // Create a stream and initialize it
    hipStream_t filterStream;
    CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

    // Alloc space on the device
    float *device_real, *device_imag;
    CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
    CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

    // Start timing for transfer down
    CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

    // Here is where we copy matrices down to the device 
    CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));

    // Stop timing for transfer down
    CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
    CUDA_ERROR_CHECK(hipEventSynchronize(stop));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

    // Start timing for the execution
    CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

    //----------------------------------------------------------------
    // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
    //        HERE BETWEEN THE CALLS FOR STARTING AND
    //        FINISHING TIMING FOR THE EXECUTION PHASE
    // BEGIN ADD KERNEL CALLS
    //----------------------------------------------------------------

    // This is an example kernel call, you should feel free to create
    // as many kernel calls as you feel are needed for your program
    // Each of the parameters are as follows:
    //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
    //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
    //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
    //    4. Stream to execute kernel on, should always be 'filterStream'
    //
    // Also note that you pass the pointers to the device memory to the kernel call
    fftKernel<<<SIZEX,SIZEY,0,filterStream>>>(device_real, device_imag, DIRECTION_X);
    fftKernel<<<SIZEY,SIZEX,0,filterStream>>>(device_real, device_imag, DIRECTION_Y);

    ifftKernel<<<SIZEX,SIZEY,0,filterStream>>>(device_real, device_imag, DIRECTION_X);
    ifftKernel<<<SIZEY,SIZEX,0,filterStream>>>(device_real, device_imag, DIRECTION_Y);
    //---------------------------------------------------------------- 
    // END ADD KERNEL CALLS
    //----------------------------------------------------------------

    // Finish timimg for the execution 
    CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
    CUDA_ERROR_CHECK(hipEventSynchronize(stop));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

    // Start timing for the transfer up
    CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

    // Here is where we copy matrices back from the device 
    CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
    CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

    // Finish timing for transfer up
    CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
    CUDA_ERROR_CHECK(hipEventSynchronize(stop));
    CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

    // Synchronize the stream
    CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
    // Destroy the stream
    CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
    // Destroy the events
    CUDA_ERROR_CHECK(hipEventDestroy(start));
    CUDA_ERROR_CHECK(hipEventDestroy(stop));

    // Free the memory
    CUDA_ERROR_CHECK(hipFree(device_real));
    CUDA_ERROR_CHECK(hipFree(device_imag));

    // Dump some usage statistics
    printf("CUDA IMPLEMENTATION STATISTICS:\n");
    printf("  Host to Device Transfer Time: %f ms\n", transferDown);
    printf("  Kernel(s) Execution Time: %f ms\n", execution);
    printf("  Device to Host Transfer Time: %f ms\n", transferUp);
    float totalTime = transferDown + execution + transferUp;
    printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
    // Return the total time to transfer and execute
    return totalTime;
}

