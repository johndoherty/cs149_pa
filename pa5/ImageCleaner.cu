#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
  // Currently does nothing
}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);

  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

